#include "hip/hip_runtime.h"
﻿#include "hip/hip_runtime.h"
#include ""
#include <hip/hip_runtime.h>
#include <hiprand.h>
#include <hiprand/hiprand_kernel.h>
#include <hipblas.h>

#include <iostream>

#include "Watermarking_CUDA.h"

// Cuda error handling
#define CUDA_CALL(x) do { if((x) != hipSuccess) { \
    printf("Error at %s:%d -- %s\n",__FILE__,__LINE__, hipGetErrorString(x)); \
    return;}} while(0)

#define CURAND_CALL(x) do { if((x)!=HIPRAND_STATUS_SUCCESS) { \
    printf("Error at %s:%d\n",__FILE__,__LINE__);\
    return;}} while(0)

#define CUBLAS_CALL(x) do { if((x)!=HIPBLAS_STATUS_SUCCESS) { \
    printf("Error at %s:%d\n",__FILE__,__LINE__);\
    return;}} while(0)

// Calculates hiprandState.
//
// state - state to be generated
// seed - unified state
// N - number of random states generated
__global__ void setup_kernel_for_random(hiprandState* state, unsigned long seed, int N)
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if(idx < N)
    hiprand_init(seed, idx, 0, &state[idx]);
}

// Calculates random values in the device
//
// global_state - global state used for generation
// random values - array containing the random values
__global__ void generate_for_random(hiprandState* global_state, float* random_values)
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    hiprandState localState = global_state[idx];
    float random = hiprand_uniform(&localState);
    random_values[idx] = random;
    global_state[idx] = localState;
}

// Adds a little to each pixel
__global__ void add_watermark(float* data, int N)
{
    //extern __shared__ int sdata[];
    
    int tidx = threadIdx.x;
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    //sdata[tidx] = data[idx];
    //__syncthreads();

   /* for (unsigned int i = blockDim.x / 2; i > 0; i >>= 1)
    {
        if (tidx < i)
        {

            data[tidx] += 0.5f;
        }
        __syncthreads();
    }*/

    if (idx < N)
        data[idx] += 0.5f;
}

// Calculates random values on the device and copies to the host,
// using hiprand Host API. (curandGenerator)
//
// host_data - copies random values to this container
// N - number of random values generated
void CalcRandWithHostAPI(float* host_data, int N)
{
    float *dev_data;
    hiprandGenerator_t generator;

    CUDA_CALL(hipMalloc((void**)&dev_data, N * sizeof(float)));

    CURAND_CALL(hiprandCreateGenerator(&generator, HIPRAND_RNG_PSEUDO_DEFAULT));
    CURAND_CALL(hiprandSetPseudoRandomGeneratorSeed(generator, 1234ULL));
    CURAND_CALL(hiprandGenerateUniform(generator, dev_data, N));

    CUDA_CALL(hipDeviceSynchronize());
    CUDA_CALL(hipMemcpy(host_data, dev_data, N * sizeof(*host_data), hipMemcpyDeviceToHost));

    // Cleanup
    CURAND_CALL(hiprandDestroyGenerator(generator));
    CUDA_CALL(hipFree(dev_data)); 
}

// Calculates random values on the device and copies to the host,
// using hiprand Device API. (hiprandState)
//
// host_data - copies random values to this container
// N - number of random values generated
void CalcRandWithDevAPI(float* host_data, int N)
{
    // 2D Thread blocks
    dim3 threads;
    if (N > 1024)
    {
        threads = dim3(512, 1);
    }
    else
    {
        threads = dim3(N, 1);
    }

    int blocks_count = floor(N / threads.x);
    dim3 blocks = dim3(blocks_count, 1);

    hiprandState* dev_states;
    float* dev_random_values;

    CUDA_CALL(hipMalloc(&dev_states, N * sizeof(hiprandState)));
    CUDA_CALL(hipMalloc(&dev_random_values, N * sizeof(*host_data)));

    // Setting up the random state
    setup_kernel_for_random << <blocks, threads >> > (dev_states, time(NULL), N);

    // Generating random numbers
    generate_for_random << <blocks, threads >> > (dev_states, dev_random_values);

    CUDA_CALL(hipMemcpy(host_data, dev_random_values, N * sizeof(*host_data), hipMemcpyDeviceToHost));

    // Cleanup
    CUDA_CALL(hipFree(dev_states));
    CUDA_CALL(hipFree(dev_random_values));
}

// Calculates the watermark in the mos significant blocks
//
// host_data - the input container with values
// N - size of the input container
// w - the watermark
// alpha - allpha for embedding the watermark
void CalcWatermark(float* host_data, int N, float w, float alpha) 
{
    float *dev_data, *temp_data;
    int max_index = 0;
    hipblasHandle_t my_handle;
    CUBLAS_CALL(hipblasCreate(&my_handle));
    
    // Allocate temporary for max
    temp_data = new float[N * sizeof(float)];
    memcpy(temp_data, host_data, N * sizeof(float));

    CUDA_CALL(hipMalloc((void**)&dev_data, N * sizeof(float)));

    CUDA_CALL(hipMemcpy(dev_data, host_data, N * sizeof(*host_data), hipMemcpyHostToDevice));
    CUBLAS_CALL(hipblasIsamax(my_handle, N, dev_data, 1, &max_index));
    temp_data[max_index] = 0;
    //CUDA_CALL(hipFree(dev_data));

    CUDA_CALL(hipMemcpy(dev_data, temp_data, N * sizeof(*host_data), hipMemcpyHostToDevice));
    CUBLAS_CALL(hipblasIsamax(my_handle, N, dev_data, 1, &max_index));
    host_data[max_index] = host_data[max_index] + alpha * w;
    CUDA_CALL(hipFree(dev_data));
}